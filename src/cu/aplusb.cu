#include <libgpu/cuda/cu/opencl_translator.cu>

#include "../cl/aplusb.cl"

void cuda_aplusb(const gpu::WorkSize &workSize, hipStream_t stream,
                 const float* a, const float* b, float* c, unsigned int n) {
    aplusb<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a, b, c, n);
    CUDA_CHECK_KERNEL(stream);
}

